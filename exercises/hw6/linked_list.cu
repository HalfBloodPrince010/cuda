
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

struct list_elem {
  int key;
  list_elem *next;
};

template <typename T>
void alloc_bytes(T &ptr, size_t num_bytes){
  hipMallocManaged(&ptr, num_bytes);
  //ptr = (T)malloc(num_bytes);
}

__host__ __device__
void print_element(list_elem *list, int ele_num, char* device){
  list_elem *elem = list;
  printf("From device: %s\n",  device);
  for (int i = 0; i < ele_num; i++) {
    printf("key = %d\n", elem->key);
    elem = elem->next;
   }
}

__global__ void gpu_print_element(list_elem *list, int ele_num){
    char device[] = "[gpu]";
    print_element(list, ele_num, device);
}

const int num_elem = 5;
const int ele = 3;
int main(){
  // list_base = ptr to head node
  list_elem *list_base, *list;
  alloc_bytes(list_base, sizeof(list_elem));
  list = list_base;
  for (int i = 0; i < num_elem; i++){
    list->key = i;
    alloc_bytes(list->next, sizeof(list_elem));
    list = list->next;
  }
  char device[] = "[cpu]";
  print_element(list_base, ele, device);
  gpu_print_element<<<1,1>>>(list_base, ele);
  hipDeviceSynchronize();
  cudaCheckErrors("cuda error!");
}
