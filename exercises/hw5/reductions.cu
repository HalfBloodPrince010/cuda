

#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t N = 2*8ULL*1024ULL*1024ULL;  // data size
//const size_t N = 256*640; // data size
const int BLOCK_SIZE = 256;  // CUDA maximum is 1024
// naive atomic reduction kernel
__global__ void atomic_red(const float *gdata, float *out){
  size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < N) atomicAdd(out, gdata[idx]);
}

__global__ void reduce(float *gdata, float *out){
     __shared__ float sdata[BLOCK_SIZE];
     int tid = threadIdx.x;
     sdata[tid] = 0.0f;
     size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

     while (idx < N) {  // grid stride loop to load data
        sdata[tid] += gdata[idx];
        idx += gridDim.x*blockDim.x;
        }

     for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        __syncthreads();
        if (tid < s)  // parallel sweep reduction
            sdata[tid] += sdata[tid + s];
        }
     if (tid == 0) out[blockIdx.x] = sdata[0];
  }

 __global__ void reduce_a(float *gdata, float *out){
     __shared__ float sdata[BLOCK_SIZE];
     int tid = threadIdx.x;
     sdata[tid] = 0.0f;
     size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

     while (idx < N) {  // grid stride loop to load data
        sdata[tid] += gdata[idx];
        idx += gridDim.x*blockDim.x;
        }

     for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        __syncthreads();
        if (tid < s)  // parallel sweep reduction
            sdata[tid] += sdata[tid + s];
        }
     if (tid == 0) atomicAdd(out, sdata[0]);
  }


__global__ void reduce_warp_shuffle(float *gdata, float *out) {
        // Each Warp Produces a value. Utmost 1024/32 warps = 32 warps
        __shared__ float sdata[32];
        int tid = threadIdx.x;
        int idx = threadIdx.x + blockDim.x * blockIdx.x;

        // each thread has a val
        float val = 0.0f;
        unsigned mask = 0xFFFFFFFFU;
        int lane = threadIdx.x % warpSize;
        int warpID = threadIdx.x / warpSize;

        //grid stride loop to handle variable length data
        while(idx < N) {
            val += gdata[idx];
            idx += gridDim.x * blockDim.x;  // Total threads in the grid
        }

        // 1st warp-shuffle operation, where each warp produces a value (tid/lane = 0)
        // holds that value
        for(int offset = warpSize/2; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(mask, val, offset);
        }
        if(lane == 0) sdata[warpID] = val;
        __syncthreads();

        // Now we have to add 32 elements, hence we just need 1 warp
        if(warpID == 0) {
            // this condition ensures we only consider the warp that participated.
            // For instance if there were only 512 threads, then 512/32 = 16
            // Only 16 warps produced the value. Hence we don't consider other 16 warps for addition.
            val = (tid < blockDim.x/warpSize)?sdata[lane]:0;

            // Final Warp Shuffle
            for(int offset = warpSize/2; offset > 0; offset >>= 1){
                val += __shfl_down_sync(mask, val, offset);
            }

            // Produced 1 value for block
            // All block after final warp shuffle atomically adds to the out.
            // This prevents another reduction.
            if(tid == 0) atomicAdd(out, val);
        }
    }


int main(){

  float *h_A, *h_sum, *d_A, *d_sum;
  h_A = new float[N];  // allocate space for data in host memory
  h_sum = new float;
  for (int i = 0; i < N; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  hipMalloc(&d_A, N*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sum, sizeof(float));  // allocate device space for sum
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, N*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  atomic_red<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("atomic reduction kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("atomic reduction kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("atomic sum reduction incorrect!\n"); return -1;}
  printf("atomic sum reduction correct!\n");
  const int blocks = 640;
  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce_a<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("reduction w/atomic kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction w/atomic kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("reduction w/atomic sum incorrect!\n"); return -1;}
  printf("reduction w/atomic sum correct!\n");
  hipMemset(d_sum, 0, sizeof(float));
  cudaCheckErrors("hipMemset failure");
  //cuda processing sequence step 1 is complete
  reduce_warp_shuffle<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
  cudaCheckErrors("reduction warp shuffle kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("reduction warp shuffle kernel execution failure or hipMemcpy H2D failure");
  if (*h_sum != (float)N) {printf("reduction warp shuffle sum incorrect!\n"); return -1;}
  printf("reduction warp shuffle sum correct!\n");
  return 0;
}
